#include "hip/hip_runtime.h"
#include "init.h"

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>



#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

using namespace std;

void TestCudaSolve(int *h_csrRowPtrA, int *h_csrColIndA, double *h_csrValA, int n, int nnz, double *h_b, double *h_x) {

    hipsolverSpHandle_t handle;
    hipsolverStatus_t status;
    hipsparseStatus_t status2;

    status = hipsolverSpCreate(&handle);


    hipsparseMatDescr_t descr;
    status2 = hipsparseCreateMatDescr(&descr);



    double* d_csrValA, *d_b, *d_x;
    int* d_csrRowPtrA, *d_csrColIndA;
    hipMalloc((void**)&d_csrValA, nnz * sizeof(double));
    hipMalloc((void**)&d_b, n * sizeof(double));
    hipMalloc((void**)&d_x, n * sizeof(double));
    hipMalloc((void**)&d_csrRowPtrA, (n + 1) * sizeof(int));
    hipMalloc((void**)&d_csrColIndA, nnz * sizeof(int));


    hipMemcpy(d_csrValA, h_csrValA, nnz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_csrRowPtrA, h_csrRowPtrA, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIndA, h_csrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(double), hipMemcpyHostToDevice);



    cout<<"start solving...\n";
    double tol = 1e-16;
    int reorder = 1;
    int singularity = 0;
    status = hipsolverSpDcsrlsvqr(handle, n, nnz, descr, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_b, tol,
                     reorder, d_x, &singularity);

    cout<<"end solving...\n";
    hipMemcpy(h_x, d_x, n * sizeof(double), hipMemcpyDeviceToHost);
    //cout<<"singularity = "<<singularity<<"\n";


    hipFree(d_csrValA);
    hipFree(d_csrRowPtrA);
    hipFree(d_csrColIndA);
    hipFree(d_b);
    hipFree(d_x);
    hipsolverSpDestroy(handle);


}

void SortCOO(int n, int nnz) {

}



void setUpDescriptor(hipsparseMatDescr_t& descrA, hipsparseMatrixType_t matrixType, hipsparseIndexBase_t indexBase) {
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, matrixType);
    hipsparseSetMatIndexBase(descrA, indexBase);
}


void setUpDescriptorLU(hipsparseMatDescr_t& descrLU, hipsparseMatrixType_t matrixType, hipsparseIndexBase_t indexBase, hipsparseFillMode_t fillMode, hipsparseDiagType_t diagType) {
    hipsparseCreateMatDescr(&descrLU);
    hipsparseSetMatType(descrLU, matrixType);
    hipsparseSetMatIndexBase(descrLU, indexBase);
    hipsparseSetMatFillMode(descrLU, fillMode);
    hipsparseSetMatDiagType(descrLU, diagType);
}


void memoryQueryLU(csrilu02Info_t& info_A, csrsv2Info_t& info_L, csrsv2Info_t& info_U, hipsparseHandle_t handle, const int N, const int nnz, hipsparseMatDescr_t descrA, hipsparseMatDescr_t descr_L,
    hipsparseMatDescr_t descr_U, double* d_A, int* d_A_RowIndices, int* d_A_ColIndices, hipsparseOperation_t matrixOperation, void** pBuffer) {

    hipsparseCreateCsrilu02Info(&info_A);
    hipsparseCreateCsrsv2Info(&info_L);
    hipsparseCreateCsrsv2Info(&info_U);

    int pBufferSize_M, pBufferSize_L, pBufferSize_U;
    hipsparseDcsrilu02_bufferSize(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, &pBufferSize_M);
    hipsparseDcsrsv2_bufferSize(handle, matrixOperation, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, &pBufferSize_L);
    hipsparseDcsrsv2_bufferSize(handle, matrixOperation, N, nnz, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, &pBufferSize_U);

    int pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));
    hipMalloc((void**)pBuffer, pBufferSize);

}


// ANALYSIS FUNCTION FOR LU DECOMPOSITION
void analysisLUDecomposition(csrilu02Info_t& info_A, csrsv2Info_t& info_L, csrsv2Info_t& info_U, hipsparseHandle_t handle, const int N, const int nnz, hipsparseMatDescr_t descrA, hipsparseMatDescr_t descr_L,
    hipsparseMatDescr_t descr_U, double* d_A, int* d_A_RowIndices, int* d_A_ColIndices, hipsparseOperation_t matrixOperation, hipsparseSolvePolicy_t solvePolicy1, hipsparseSolvePolicy_t solvePolicy2, void* pBuffer) {

    int structural_zero;

    hipsparseDcsrilu02_analysis(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, solvePolicy1, pBuffer);
    hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(handle, info_A, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status) { printf("A(%d,%d) is missing\n", structural_zero, structural_zero); }

    hipsparseDcsrsv2_analysis(handle, matrixOperation, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, solvePolicy1, pBuffer);
    hipsparseDcsrsv2_analysis(handle, matrixOperation, N, nnz, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, solvePolicy2, pBuffer);

}


// COMPUTE LU DECOMPOSITION FOR SPARSE MATRICES

void computeSparseLU(csrilu02Info_t& info_A, hipsparseHandle_t handle, const int N, const int nnz, hipsparseMatDescr_t descrA, double* d_A, int* d_A_RowIndices,
    int* d_A_ColIndices, hipsparseSolvePolicy_t solutionPolicy, void* pBuffer) {

    int numerical_zero;

    hipsparseDcsrilu02(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, solutionPolicy, pBuffer);
    hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(handle, info_A, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status) { printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero); }

}


void Prepare_CSR(int *rows, int *ptr, int nnz, int n) {
    hipsparseHandle_t    handle;
    hipsparseCreate(&handle);

    int *d_rows;
    hipMalloc(&d_rows, nnz * sizeof(int));
    hipMemcpy(d_rows, rows, nnz * sizeof(int), hipMemcpyHostToDevice);

    int *d_ptr;
    hipMalloc(&d_ptr, (n + 1) * sizeof(int));
    hipMemcpy(d_ptr, ptr, (n + 1) * sizeof(int), hipMemcpyHostToDevice);

    hipsparseXcoo2csr(handle, d_rows, nnz, n, d_ptr, HIPSPARSE_INDEX_BASE_ZERO);

    hipMemcpy(ptr, d_ptr, (n + 1) * sizeof(int), hipMemcpyDeviceToHost);
}

void LU_GPU_SOLVE(int *h_A_RowIndices, int *h_A_ColIndices, double *h_A, int n, int nnz, double *h_x, double *result)
{

    hipsparseHandle_t    handle;

    hipsparseMatDescr_t  descrA = 0;
    hipsparseMatDescr_t  descr_L = 0;
    hipsparseMatDescr_t  descr_U = 0;

    csrilu02Info_t      info_A = 0;
    csrsv2Info_t        info_L = 0;
    csrsv2Info_t        info_U = 0;

    void* pBuffer = 0;

    hipEvent_t start, stop;
    hipsparseCreate(&handle);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float gpuTime = 0.0;

    hipEventRecord(start, 0);


    const int Nrows = n;
    const int Ncols = n;
    const int N = Nrows;


    double* d_x;
    hipMalloc(&d_x, Nrows * sizeof(double));
    hipMemcpy(d_x, h_x, Nrows * sizeof(double), hipMemcpyHostToDevice);



    setUpDescriptor(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ONE);

//    for (int i = 0; i < nnz; i++) {
//        printf("%f ", h_A[i]);
//    }


    for (int i = 0; i < nnz; i++) {
        //h_A_ColIndices[i]++;
        //printf("%d ", h_A_ColIndices[i]);
    }


    double* d_A;
    hipMalloc(&d_A, nnz * sizeof(*d_A));

    int* d_A_RowIndices;
    hipMalloc(&d_A_RowIndices, (Nrows + 1) * sizeof(*d_A_RowIndices));

    int* d_A_ColIndices;
    hipMalloc(&d_A_ColIndices, nnz * sizeof(*d_A_ColIndices));

    hipMemcpy(d_A, h_A, nnz * sizeof(*h_A), hipMemcpyHostToDevice);
    hipMemcpy(d_A_RowIndices, h_A_RowIndices, (Nrows + 1) * sizeof(*h_A_RowIndices), hipMemcpyHostToDevice);
    hipMemcpy(d_A_ColIndices, h_A_ColIndices, nnz * sizeof(*h_A_ColIndices), hipMemcpyHostToDevice);


    hipMemcpy(h_A, d_A, nnz * sizeof(*h_A), hipMemcpyDeviceToHost);

    setUpDescriptorLU(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ONE, HIPSPARSE_FILL_MODE_LOWER, HIPSPARSE_DIAG_TYPE_UNIT);
    setUpDescriptorLU(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ONE, HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);


    memoryQueryLU(info_A, info_L, info_U, handle, N, nnz, descrA, descr_L, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, HIPSPARSE_OPERATION_NON_TRANSPOSE, &pBuffer);


    analysisLUDecomposition(info_A, info_L, info_U, handle, N, nnz, descrA, descr_L, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_SOLVE_POLICY_NO_LEVEL,HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);


    computeSparseLU(info_A, handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);


    double* d_z;
    hipMalloc(&d_z, N * sizeof(double));

    const double alpha = 1.;
    hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, &alpha, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, d_x, d_z, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);


    double* d_y;
    hipMalloc(&d_y, N * sizeof(double));

    hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, &alpha, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, d_z, d_y, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);


    double* h_y = (double*)malloc(Ncols * sizeof(double));
    hipMemcpy(h_x, d_y, N * sizeof(double), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("GPU time = %.4f \n", gpuTime);

    printf("\n\nFinal result\n");
    for (int i = 0; i < N; i++) {
        printf("%f ", h_x[i]);
    }
}




///////////////////////////////////////////////////////
__global__
void cgm_gpu(double *z_k, double *r_k, double *Az,
             int* x, int *y, double *data, double *b,
             double *x_k, const int n, int sparse_size, double *partialSum) {

    double mf = 0.0, alpha, beta, eps = 0.00001, Spz, Spr, Spr1;

    int tx = threadIdx.x;
    int i = tx + blockIdx.x * blockDim.x;

    if (i < n) {
        partialSum[tx] = b[i] * b[i];
    }

    int stride;
    for (stride = blockDim.x/2; stride > 0;  stride >>= 1) {
        __syncthreads();
        if (tx < stride) {
            partialSum[tx] += partialSum[tx + stride];
        }
    }

    if (tx == 0) {
        //b[blockIdx.x] = partialSum[tx];
        mf = partialSum[tx];
    }



    x_k[i] = 0.2;
    Az[i] = 0.0;

    if (i < sparse_size) {
        Az[x[i]] += data[i] * x_k[y[i]];
    }
    r_k[i] = b[i] - Az[i];
    z_k[i] = r_k[i];

    //do{
        Spz=0.0;
        Spr=0.0;
        Az[i] = 0.0;
        if (i < sparse_size) {
            Az[x[i]] += data[i] * z_k[y[i]];
        }
        //Spz
        if (i < n) {
            partialSum[tx] = Az[i] * z_k[i];
        }

        for (stride = blockDim.x/2; stride > 0;  stride >>= 1) {
            __syncthreads();
            if (tx < stride) {
                partialSum[tx] += partialSum[tx + stride];
            }
        }
        if (tx == 0) {
            //b[blockIdx.x] = partialSum[tx];
            Spz = partialSum[tx];
            printf("Spz=%f\n", Spz);
        }
        //Spr
        if (i < n) {
            partialSum[tx] = r_k[i] * r_k[i];
        }

        for (stride = blockDim.x/2; stride > 0;  stride >>= 1) {
            __syncthreads();
            if (tx < stride) {
                partialSum[tx] += partialSum[tx + stride];
            }
        }

        if (tx == 0) {
            //b[blockIdx.x] = partialSum[tx];
            Spr = partialSum[tx];
            printf("Spr=%f\n", Spr);
        }
        ////
        alpha = Spr / Spz;
        Spr1 = 0.0;
        x_k[i] += alpha * z_k[i];
        r_k[i] -= alpha * Az[i];
        //Spr1
        if (i < n) {
            partialSum[tx] = r_k[i] * r_k[i];
            printf("%f ", r_k[i]);
        }

        for (stride = blockDim.x/2; stride > 0;  stride >>= 1) {
            __syncthreads();
            if (tx < stride) {
                partialSum[tx] += partialSum[tx + stride];
            }
        }

        if (tx == 0) {
            //b[blockIdx.x] = partialSum[tx];
            Spr1 = partialSum[tx];
            printf("Spr1=%f\n", Spr1);
        }
        ////
        beta = Spr1 / Spr;
        z_k[i] = r_k[i] + beta * z_k[i];


   // } while (Spr1 / mf > eps * eps);

    if (i == 0)
        printf("GPU CGM SUCCESS\n");
}

void callCGM_GPU(int *x, int *y, double *data, double *b, double *x_k, int n, int sparse_size) {
    double *z_k, *r_k, *Az;
    double *d_z_k, *d_r_k, *d_Az, *d_data, *d_b, *d_x_k, *partialSum;
    int *d_x, *d_y;
    z_k = (double*)malloc(n * sizeof(double));
    r_k = (double*)malloc(n * sizeof(double));
    Az = (double*)malloc(n * sizeof(double));


    hipMalloc(&d_z_k, n * sizeof(double));
    hipMalloc(&d_r_k, n * sizeof(double));
    hipMalloc(&d_Az, n * sizeof(double));
    hipMalloc(&d_x, sparse_size * sizeof(int));
    hipMalloc(&d_y, sparse_size * sizeof(int));
    hipMalloc(&d_data, sparse_size * sizeof(double));
    hipMalloc(&d_b, n * sizeof(double));
    hipMalloc(&d_x_k, n * sizeof(double));
    hipMalloc(&partialSum, n * sizeof(double));

    hipMemcpy(d_z_k, z_k, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_r_k, r_k, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Az, Az, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, sparse_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sparse_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_data, data, sparse_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x_k, x_k, n * sizeof(double), hipMemcpyHostToDevice);

    cgm_gpu<<<1, n>>>(d_z_k, d_r_k, d_Az, d_x, d_y, d_data, d_b, d_x_k, n, sparse_size, partialSum);

    hipMemcpy(x_k, d_x_k, n * sizeof(double), hipMemcpyDeviceToHost);


//    hipFree(d_x);
//    hipFree(d_y);
//    free(x);
//    free(y);
}



__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void callCudaKernel()
{
  int N = 1<<8;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = max(maxError, abs(y[i]-4.0f));
    printf("%f ", y[i]);
  }
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}

