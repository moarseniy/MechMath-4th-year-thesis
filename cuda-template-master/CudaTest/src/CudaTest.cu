#include "hip/hip_runtime.h"
#include "init.h"
#include <stdio.h>


__global__
void cgm_gpu(double *z_k, double *r_k, double *Az,
             int* x, int *y, double *data, double *b,
             double *x_k, const int n, int sparse_size, double *partialSum) {

    double mf = 0.0, alpha, beta, eps = 0.00001, Spz, Spr, Spr1;

    int tx = threadIdx.x;
    int i = tx + blockIdx.x * blockDim.x;

    if (i < n) {
        partialSum[tx] = b[i] * b[i];
    }

    int stride;
    for (stride = blockDim.x/2; stride > 0;  stride >>= 1) {
        __syncthreads();
        if (tx < stride) {
            partialSum[tx] += partialSum[tx + stride];
        }
    }

    if (tx == 0) {
        //b[blockIdx.x] = partialSum[tx];
        mf = partialSum[tx];
    }



    x_k[i] = 0.2;
    Az[i] = 0.0;

    if (i < sparse_size) {
        Az[x[i]] += data[i] * x_k[y[i]];
    }
    r_k[i] = b[i] - Az[i];
    z_k[i] = r_k[i];

    //do{
        Spz=0.0;
        Spr=0.0;
        Az[i] = 0.0;
        if (i < sparse_size) {
            Az[x[i]] += data[i] * z_k[y[i]];
        }
        //Spz
        if (i < n) {
            partialSum[tx] = Az[i] * z_k[i];
        }

        for (stride = blockDim.x/2; stride > 0;  stride >>= 1) {
            __syncthreads();
            if (tx < stride) {
                partialSum[tx] += partialSum[tx + stride];
            }
        }
        if (tx == 0) {
            //b[blockIdx.x] = partialSum[tx];
            Spz = partialSum[tx];
            printf("Spz=%f\n", Spz);
        }
        //Spr
        if (i < n) {
            partialSum[tx] = r_k[i] * r_k[i];
        }

        for (stride = blockDim.x/2; stride > 0;  stride >>= 1) {
            __syncthreads();
            if (tx < stride) {
                partialSum[tx] += partialSum[tx + stride];
            }
        }

        if (tx == 0) {
            //b[blockIdx.x] = partialSum[tx];
            Spr = partialSum[tx];
            printf("Spr=%f\n", Spr);
        }
        ////
        alpha = Spr / Spz;
        Spr1 = 0.0;
        x_k[i] += alpha * z_k[i];
        r_k[i] -= alpha * Az[i];
        //Spr1
        if (i < n) {
            partialSum[tx] = r_k[i] * r_k[i];
            printf("%f ", r_k[i]);
        }

        for (stride = blockDim.x/2; stride > 0;  stride >>= 1) {
            __syncthreads();
            if (tx < stride) {
                partialSum[tx] += partialSum[tx + stride];
            }
        }

        if (tx == 0) {
            //b[blockIdx.x] = partialSum[tx];
            Spr1 = partialSum[tx];
            printf("Spr1=%f\n", Spr1);
        }
        ////
        beta = Spr1 / Spr;
        z_k[i] = r_k[i] + beta * z_k[i];


   // } while (Spr1 / mf > eps * eps);

    if (i == 0)
        printf("GPU CGM SUCCESS\n");
}

void callCGM_GPU(int *x, int *y, double *data, double *b, double *x_k, int n, int sparse_size) {
    double *z_k, *r_k, *Az;
    double *d_z_k, *d_r_k, *d_Az, *d_data, *d_b, *d_x_k, *partialSum;
    int *d_x, *d_y;
    z_k = (double*)malloc(n * sizeof(double));
    r_k = (double*)malloc(n * sizeof(double));
    Az = (double*)malloc(n * sizeof(double));


    hipMalloc(&d_z_k, n * sizeof(double));
    hipMalloc(&d_r_k, n * sizeof(double));
    hipMalloc(&d_Az, n * sizeof(double));
    hipMalloc(&d_x, sparse_size * sizeof(int));
    hipMalloc(&d_y, sparse_size * sizeof(int));
    hipMalloc(&d_data, sparse_size * sizeof(double));
    hipMalloc(&d_b, n * sizeof(double));
    hipMalloc(&d_x_k, n * sizeof(double));
    hipMalloc(&partialSum, n * sizeof(double));

    hipMemcpy(d_z_k, z_k, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_r_k, r_k, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Az, Az, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, sparse_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sparse_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_data, data, sparse_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x_k, x_k, n * sizeof(double), hipMemcpyHostToDevice);

    cgm_gpu<<<1, n>>>(d_z_k, d_r_k, d_Az, d_x, d_y, d_data, d_b, d_x_k, n, sparse_size, partialSum);

    hipMemcpy(x_k, d_x_k, n * sizeof(double), hipMemcpyDeviceToHost);


//    hipFree(d_x);
//    hipFree(d_y);
//    free(x);
//    free(y);
}



__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void callCudaKernel()
{
  int N = 1<<8;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = max(maxError, abs(y[i]-4.0f));
    printf("%f ", y[i]);
  }
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}

