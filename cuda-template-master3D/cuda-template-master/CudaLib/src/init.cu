#include "hip/hip_runtime.h"

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>



#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <math.h>
#include <vector>
#include "init.h"
//#include "femfunc.h"

#include "Linal2.h"

using namespace std;

__device__
float det3(float a0, float a1, float a2, float a3,
           float a4, float a5, float a6, float a7, float a8) {
    return a0*a4*a8 +
                    a1*a6*a5 +
                    a2*a3*a7 -
                    a6*a4*a2 -
                    a0*a5*a7 -
                    a1*a3*a8;
}

__device__
float det3x3(float *c) {
    return c[0]*c[4]*c[8] +
            c[1]*c[6]*c[5] +
            c[2]*c[3]*c[7] -
            c[6]*c[4]*c[2] -
            c[0]*c[5]*c[7] -
            c[1]*c[3]*c[8];
}

__device__
float det4x4(float *c) {
    float v1 = det3(c[5], c[6], c[7], c[9], c[10], c[11], c[13], c[14], c[15]);
    float v2 = det3(c[1], c[2], c[3], c[9], c[10], c[11], c[13], c[14], c[15]);
    float v3 = det3(c[1], c[2], c[3], c[5], c[6], c[7], c[13], c[14], c[15]);
    float v4 = det3(c[1], c[2], c[3], c[5], c[6], c[7], c[9], c[10], c[11]);
    return v1 - v2 + v3 - v4;
}

__device__
float det(float *c, int size) {
    if (size == 1) {
        return c[0];
    } else if (size == 2) {
        return c[0 + 0 * 2] * c[1 + 1 * 2] - c[0 + 1 * 2] * c[1 + 0 * 2];
    } else if (size == 3) {
        return c[0]*c[4]*c[8] +
                c[1]*c[6]*c[5] +
                c[2]*c[3]*c[7] -
                c[6]*c[4]*c[2] -
                c[0]*c[5]*c[7] -
                c[1]*c[3]*c[8];
    } else if (size == 4) {
        float v1 = det3(c[5], c[6], c[7], c[9], c[10], c[11], c[13], c[14], c[15]);
        float v2 = det3(c[1], c[2], c[3], c[9], c[10], c[11], c[13], c[14], c[15]);
        float v3 = det3(c[1], c[2], c[3], c[5], c[6], c[7], c[13], c[14], c[15]);
        float v4 = det3(c[1], c[2], c[3], c[5], c[6], c[7], c[9], c[10], c[11]);
        return v1 - v2 + v3 - v4;
    }
}

__device__
void Get_matrix(float *a, int n, float *c, int indRow, int indCol) {
    //float *a = (float*)malloc(3 * 3 * sizeof (float));
    int ki = 0;
    for (int i = 0; i < n; i++) {
        if (i != indRow) {
            for (int j = 0, kj = 0; j < n; j++) {
                if (j != indCol) {
                    a[kj + ki * 3] = c[j + i * n];
                    kj++;
                }
            }
            ki++;
        }
    }

    //return a;
}

__device__
void inverse(float *ic, float *b, int size) {
    //float *ic = (float*)malloc(4 * 4 * sizeof (float));
    //printf("%f", b[0]);
    float determinant = det4x4(b);

    if (determinant) {
        for (int i = 0; i < size; i++) {
            for (int j = 0; j < size; j++) {
                float *temp =(float*)malloc(3 * 3 * sizeof (float));
                //__shared__ float temp[3 * 3];
                Get_matrix(temp, size, b, i, j);
                ic[j + i * size] = ((i + j + 2) % 2 == 0 ? 1.0 : -1.0) * det3x3(temp) / determinant;
                free(temp);
            }
        }
    }

    float swap;
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            if (i > j) {
               swap = ic[j + i * size];
               ic[j + i * size] = ic[i + j * size];
               ic[i + j * size] = swap;
            }
        }
    }

   //return ic;
}

__global__
void CalculateLocalSets(int elementsCount, const float *nodesX, const float *nodesY, const float *nodesZ, int nodesCount,
                        int *elements, int k, int sumColors, float *D, int *K_x, int *K_y, float *K_value, int *constraints, int constraintCount) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("!%d ", index);
    //printf("%d ", elementsCount);

    if (index < elementsCount) {
        //__shared__ float C[4 * 4];
        //__shared__ float IC[4 * 4];

        float *C = (float*)malloc(4 * 4 * sizeof(float));
        float *IC = (float*)malloc(4 * 4 * sizeof(float));

        C[0 + 0 * 4] = C[0 + 1 * 4] = C[0 + 2 * 4] = C[0 + 3 * 4] = 1.0;
        C[1 + 0 * 4] = nodesX[elements[4 * (index + sumColors) + 0]]; C[1 + 1 * 4] = nodesX[elements[4 * (index + sumColors) + 1]]; C[1 + 2 * 4] = nodesX[elements[4 * (index + sumColors) + 2]]; C[1 + 3 * 4] = nodesX[elements[4 * (index + sumColors) + 3]];
        C[2 + 0 * 4] = nodesY[elements[4 * (index + sumColors) + 0]]; C[2 + 1 * 4] = nodesY[elements[4 * (index + sumColors) + 1]]; C[2 + 2 * 4] = nodesY[elements[4 * (index + sumColors) + 2]]; C[2 + 3 * 4] = nodesY[elements[4 * (index + sumColors) + 3]];
        C[3 + 0 * 4] = nodesZ[elements[4 * (index + sumColors) + 0]]; C[3 + 1 * 4] = nodesZ[elements[4 * (index + sumColors) + 1]]; C[3 + 2 * 4] = nodesZ[elements[4 * (index + sumColors) + 2]]; C[3 + 3 * 4] = nodesZ[elements[4 * (index + sumColors) + 3]];

        float determinant = det4x4(C);

//        printf("%d %d %d %d ", elements[4 * (index + sumColors) + 0], elements[4 * (index + sumColors) + 1],
//                elements[4 * (index + sumColors) + 2], elements[4 * (index + sumColors) + 3]);
//        printf("%d %d %d %d ", 4 * (index + sumColors) + 0, 4 * (index + sumColors) + 1,
//                4 * (index + sumColors) + 2, 4 * (index + sumColors) + 3);



        //printf("%f ", nodesX[elements[4 * (index + sumColors) + 3]]);

        //printf("%f ", C[1]);

        //printf("%d ", index);



        //__syncthreads();

        inverse(IC, C, 4);
        free(C);

        //inverse(C, IC, 4);
        //printf("%f ", IC[1]);

        //__syncthreads();

        //__shared__ float B[6 * 12];
        float *B = (float*)malloc(6 * 12 * sizeof(float));

        for (int i = 0; i < 4; i++) {
            B[(3 * i + 0) + 0 * 12] = IC[i + 1 * 4];
            B[(3 * i + 1) + 0 * 12] = 0.0;
            B[(3 * i + 2) + 0 * 12] = 0.0;

            B[(3 * i + 0) + 1 * 12] = 0.0;
            B[(3 * i + 1) + 1 * 12] = IC[i + 2 * 4];
            B[(3 * i + 2) + 1 * 12] = 0.0;

            B[(3 * i + 0) + 2 * 12] = 0.0;
            B[(3 * i + 1) + 2 * 12] = 0.0;
            B[(3 * i + 2) + 2 * 12] = IC[i + 3 * 4];

            B[(3 * i + 0) + 3 * 12] = IC[i + 2 * 4];
            B[(3 * i + 1) + 3 * 12] = IC[i + 1 * 4];
            B[(3 * i + 2) + 3 * 12] = 0.0;

            B[(3 * i + 0) + 4 * 12] = 0.0;
            B[(3 * i + 1) + 4 * 12] = IC[i + 3 * 4];
            B[(3 * i + 2) + 4 * 12] = IC[i + 2 * 4];

            B[(3 * i + 0) + 5 * 12] = IC[i + 3 * 4];
            B[(3 * i + 1) + 5 * 12] = 0.0;
            B[(3 * i + 2) + 5 * 12] = IC[i + 1 * 4];
        }

        free(IC);

        //__syncthreads();


        //printf("%d-%f ",i + sumColors, determinant);

//        for (int i = 0; i < 6; i++) {
//            for (int j = 0; j < 12; j++) {
//                printf("%f ", B[j + i * 12]);
//            }
//            //printf("\n");
//        }
        //printf("%f ", B[0]);


        //transpose
        //__shared__ float B_T[12 * 6];
        float *B_T = (float*)malloc(12 * 6 * sizeof(float));
        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 6; j++) {
                B_T[j + i * 6] = B[i + j * 12];
                //printf("%f ", B_T[j + i * 6]);
            }
            //printf("\n");
        }
        //printf("%f-%f ", B[2], B_T[2]);

        //__syncthreads();

        //product B_T * D
        float *temp = (float*)malloc(12 * 6 * sizeof(float));
        //__shared__ float temp[12 * 6];

        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 6; j++) {
                temp[j + i * 6] = 0.0;
                for (int k = 0; k < 6; k++) {
                    temp[j + i * 6] += B_T[k + i * 6] * D[j + k * 6];
                }
            }
        }

        free(B_T);

        //__syncthreads();

        //product (B_T * D) * B
        float *K = (float*)malloc(12 * 12 * sizeof(float));
        //__shared__ float K[12 * 12];

        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 12; j++) {
                K[j + i * 12] = 0.0;
                for (int k = 0; k < 6; k++) {
                    K[j + i * 12] += temp[k + i * 6] * B[j + k * 12];
                }
            }
        }

        free(temp);
        free(B);

        //__syncthreads();

        //scale K * |det(C)| / 6.0
        for (int i = 0; i < 12; i++) {
            for (int j = 0; j < 12; j++) {
                //K[j + i * 12] *= (determinant > 0 ? determinant : (-1 * determinant)) / 6.0;
                K[j + i * 12] *= (fabs(determinant)) / 6.0;
                //printf("%f ", K[j + i * 12]);
            }
        }



//        for (int i = 0; i < 12; i ++) {
//            for (int j = 0; j < 12; j++) {
//                printf("%f ", K[j + i * 12]);
//            }
//            printf("\n");
//        }

        //printf("%f ", K_value[0]);

        //__syncthreads();
        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 4; j++) {
                int idi = 3 * elements[4 * (index + sumColors) + i];
                int idj = 3 * elements[4 * (index + sumColors) + j];
                //printf("%d-%d ", idi, idj);
                K_x[(idj + 0) + (idi + 0) * 3 * nodesCount] = idi + 0;
                K_y[(idj + 0) + (idi + 0) * 3 * nodesCount] = idj + 0;
                K_value[(idj + 0) + (idi + 0) * 3 * nodesCount] += K[(3 * j + 0) + (3 * i + 0) * 12];

                K_x[(idj + 1) + (idi + 0) * 3 * nodesCount] = idi + 0;
                K_y[(idj + 1) + (idi + 0) * 3 * nodesCount] = idj + 1;
                K_value[(idj + 1) + (idi + 0) * 3 * nodesCount] += K[(3 * j + 1) + (3 * i + 0) * 12];

                K_x[(idj + 2) + (idi + 0) * 3 * nodesCount] = idi + 0;
                K_y[(idj + 2) + (idi + 0) * 3 * nodesCount] = idj + 2;
                K_value[(idj + 2) + (idi + 0) * 3 * nodesCount] += K[(3 * j + 2) + (3 * i + 0) * 12];

                K_x[(idj + 0) + (idi + 1) * 3 * nodesCount] = idi + 1;
                K_y[(idj + 0) + (idi + 1) * 3 * nodesCount] = idj + 0;
                K_value[(idj + 0) + (idi + 1) * 3 * nodesCount] += K[(3 * j + 0) + (3 * i + 1) * 12];

                K_x[(idj + 1) + (idi + 1) * 3 * nodesCount] = idi + 1;
                K_y[(idj + 1) + (idi + 1) * 3 * nodesCount] = idj + 1;
                K_value[(idj + 1) + (idi + 1) * 3 * nodesCount] += K[(3 * j + 1) + (3 * i + 1) * 12];

                K_x[(idj + 2) + (idi + 1) * 3 * nodesCount] = idi + 1;
                K_y[(idj + 2) + (idi + 1) * 3 * nodesCount] = idj + 2;
                K_value[(idj + 2) + (idi + 1) * 3 * nodesCount] += K[(3 * j + 2) + (3 * i + 1) * 12];

                K_x[(idj + 0) + (idi + 2) * 3 * nodesCount] = idi + 2;
                K_y[(idj + 0) + (idi + 2) * 3 * nodesCount] = idj + 0;
                K_value[(idj + 0) + (idi + 2) * 3 * nodesCount] += K[(3 * j + 0) + (3 * i + 2) * 12];

                K_x[(idj + 1) + (idi + 2) * 3 * nodesCount] = idi + 2;
                K_y[(idj + 1) + (idi + 2) * 3 * nodesCount] = idj + 1;
                K_value[(idj + 1) + (idi + 2) * 3 * nodesCount] += K[(3 * j + 1) + (3 * i + 2) * 12];

                K_x[(idj + 2) + (idi + 2) * 3 * nodesCount] = idi + 2;
                K_y[(idj + 2) + (idi + 2) * 3 * nodesCount] = idj + 2;
                K_value[(idj + 2) + (idi + 2) * 3 * nodesCount] += K[(3 * j + 2) + (3 * i + 2) * 12];

                for (int t = 0; t < constraintCount; t++) {
                    for (int i1 = 0; i1 < 3; i1++) {
                        for (int j1 = 0; j1 < 3; j1++) {
                            if (idi + i1 == constraints[t] || idj + j1 == constraints[t]) {
                                if (idi + i1 == idj + j1) {
                                    K_value[(idj + j1) + (idi + i1) * 3 * nodesCount] = 1.0;
                                } else {
                                    K_value[(idj + j1) + (idi + i1) * 3 * nodesCount] = 0.0;
                                }
                            }
                        }
                    }
                }
//                K_x[(idj + 0) + (idi + 0) * 3 * nodesCount] = idi + 0;
//                K_y[(idj + 0) + (idi + 0) * 3 * nodesCount] = idj + 0;
//                K_value[(idj + 0) + (idi + 0) * 3 * nodesCount] += K[(3 * j + 0) + (3 * i + 0) * 12];

//                K_x[(idj + 1) + (idi + 0) * 3 * nodesCount] = idi + 0;
//                K_y[(idj + 1) + (idi + 0) * 3 * nodesCount] = idj + 1;
//                K_value[(idj + 1) + (idi + 0) * 3 * nodesCount] += K[(3 * j + 1) + (3 * i + 0) * 12];

//                K_x[(idj + 2) + (idi + 0) * 3 * nodesCount] = idi + 0;
//                K_y[(idj + 2) + (idi + 0) * 3 * nodesCount] = idj + 2;
//                K_value[(idj + 2) + (idi + 0) * 3 * nodesCount] += K[(3 * j + 2) + (3 * i + 0) * 12];

//                K_x[(idj + 0) + (idi + 1) * 3 * nodesCount] = idi + 1;
//                K_y[(idj + 0) + (idi + 1) * 3 * nodesCount] = idj + 0;
//                K_value[(idj + 0) + (idi + 1) * 3 * nodesCount] += K[(3 * j + 0) + (3 * i + 1) * 12];

//                K_x[(idj + 1) + (idi + 1) * 3 * nodesCount] = idi + 1;
//                K_y[(idj + 1) + (idi + 1) * 3 * nodesCount] = idj + 1;
//                K_value[(idj + 1) + (idi + 1) * 3 * nodesCount] += K[(3 * j + 1) + (3 * i + 1) * 12];

//                K_x[(idj + 2) + (idi + 1) * 3 * nodesCount] = idi + 1;
//                K_y[(idj + 2) + (idi + 1) * 3 * nodesCount] = idj + 2;
//                K_value[(idj + 2) + (idi + 1) * 3 * nodesCount] += K[(3 * j + 2) + (3 * i + 1) * 12];

//                K_x[(idj + 0) + (idi + 2) * 3 * nodesCount] = idi + 2;
//                K_y[(idj + 0) + (idi + 2) * 3 * nodesCount] = idj + 0;
//                K_value[(idj + 0) + (idi + 2) * 3 * nodesCount] += K[(3 * j + 0) + (3 * i + 2) * 12];

//                K_x[(idj + 1) + (idi + 2) * 3 * nodesCount] = idi + 2;
//                K_y[(idj + 1) + (idi + 2) * 3 * nodesCount] = idj + 1;
//                K_value[(idj + 1) + (idi + 2) * 3 * nodesCount] += K[(3 * j + 1) + (3 * i + 2) * 12];

//                K_x[(idj + 2) + (idi + 2) * 3 * nodesCount] = idi + 2;
//                K_y[(idj + 2) + (idi + 2) * 3 * nodesCount] = idj + 2;
//                K_value[(idj + 2) + (idi + 2) * 3 * nodesCount] += K[(3 * j + 2) + (3 * i + 2) * 12];

            }
            //__syncthreads();
        }

        free(K);
        //__syncthreads();

    }
    //__syncthreads();

}

__global__
void ApplyConstraintsCuda(int *K_x, int *K_y, float *K_value, int *constraints, int constraintsCount, int i) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < constraintsCount) {
        //printf("[%d]%d ",index, constraints[index]);
        if (K_x[i] == constraints[index] || K_y[i] == constraints[index]) {
            if (K_x[i] == K_y[i]) {
                K_value[i] = 1.0;
            } else {
                K_value[i] = 0.0;
            }
        }
    }
    __syncthreads();
}

__global__
void CountNonZeroValues(int *K_x, int *K_y, float *K_value, int SIZE, int *nnz) {
    nnz[0] = 0;
    float epsilon = 1e-50;
    for (int i = 0; i < SIZE; i++) {
        if (K_value[i] != 0.0) {
            //printf("%f ", K_value[i]);
            nnz[0]++;
        }
    }
    //printf("(func)%d %d\n", nnz[0], elementsCount);
    //__syncthreads();
}

__global__
void ConvertToCSR(int *K_x, int *K_y, float *K_value, int *ptr, int *ind, float *data, int SIZE, int nodesCount) {
    int k = 0;
    float epsilon = 1e-50;

    for (int i = 0; i < 3 * nodesCount + 1; i++) {
        ptr[i] = 0;
    }

    for (int i = 0; i < SIZE; i++) {
        if (K_value[i] != 0.0) {
            data[k] = K_value[i];
            ind[k] = K_y[i];
            ptr[K_x[i] + 1]++;
            k++;
        }
    }
    printf("k=%d\n", k);

    for (int i = 0; i < 3 * nodesCount; i++) {
        ptr[i + 1] += ptr[i];
    }
    printf("ConvertSuccess\n");
    __syncthreads();
}

__global__
void TestCudaFunc(int a) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < a) {
        printf("%d ", index);
    }
}

void FiniteElementMethodCUDA(float *h_D, int *h_elements,
                             int *h_elements0, int *h_elements1, int *h_elements2, int *h_elements3, int elementsCount,
                             float *h_nodesX, float *h_nodesY, float *h_nodesZ, int nodesCount,
                             int *h_colors, int colorsCount, int *h_constraints, int constraintsCount, float *h_b) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float gpuTime = 0.0;
    hipEventRecord(start, 0);

    int *d_elements, *d_elements0, *d_elements1, *d_elements2, *d_elements3, *d_colors, *d_K_x, *d_K_y, *d_constraints;
    float *d_D, *d_nodesX, *d_nodesY, *d_nodesZ,  *d_K_value, *d_b, *d_x;

    int SIZE = 3 * nodesCount * 3 * nodesCount;

    int *d_nnz;
    int *h_nnz = new int[1];
    hipMalloc((void**)&d_nnz, 1 * sizeof(int));

    int *h_K_x = new int[SIZE];
    int *h_K_y = new int[SIZE];
    float *h_K_value = new float[SIZE];

    hipMalloc((void**)&d_b, 3 * nodesCount * sizeof(float));
    hipMemcpy(d_b, h_b, 3 * nodesCount * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_x, 3 * nodesCount * sizeof(float));

    //cout << "elementsCount = " << elementsCount << endl;
    hipMalloc((void**)&d_elements, 4 * elementsCount * sizeof(int));
    hipMemcpy(d_elements, h_elements, 4 * elementsCount * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_D, 6 * 6 * sizeof(float));
    hipMemcpy(d_D, h_D, 6 * 6 * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_K_x, SIZE * sizeof(int));
    hipMalloc((void**)&d_K_y, SIZE * sizeof(int));
    hipMalloc((void**)&d_K_value, SIZE * sizeof(float));

//    hipMalloc((void**)&d_elements0, elementsCount * sizeof(int));
//    hipMalloc((void**)&d_elements1, elementsCount * sizeof(int));
//    hipMalloc((void**)&d_elements2, elementsCount * sizeof(int));
//    hipMalloc((void**)&d_elements3, elementsCount * sizeof(int));
//    hipMemcpy(d_elements0, h_elements0, elementsCount * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(d_elements1, h_elements1, elementsCount * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(d_elements2, h_elements2, elementsCount * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(d_elements3, h_elements3, elementsCount * sizeof(int), hipMemcpyHostToDevice);


    hipMalloc((void**)&d_nodesX, nodesCount * sizeof(float));
    hipMalloc((void**)&d_nodesY, nodesCount * sizeof(float));
    hipMalloc((void**)&d_nodesZ, nodesCount * sizeof(float));
    hipMemcpy(d_nodesX, h_nodesX, nodesCount * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_nodesY, h_nodesY, nodesCount * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_nodesZ, h_nodesZ, nodesCount * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_colors, colorsCount * sizeof(int));
    hipMemcpy(d_colors, h_colors, colorsCount * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_constraints, constraintsCount * sizeof(int));
    hipMemcpy(d_constraints, h_constraints, constraintsCount * sizeof(int), hipMemcpyHostToDevice);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("GPU(Read data) = %.4f ms\n", gpuTime);

    ////////
//    int size = 1000, temp_sum = 5;
//    int *test1 = new int[size];

//    for (int k = 0; k < 10; k++) {
        //TestCudaFunc<<<1, 50>>>(temp_sum);
//        temp_sum += 2;

        //hipDeviceSynchronize();
        //cout << endl;
//    }

    ///////

    int sumColors = 0;
    cout << "\ncolorsCount = " << colorsCount << endl;
    for (int k = 0; k < colorsCount; k++) {
        CalculateLocalSets<<<(255+h_colors[k])/256, 256>>> (h_colors[k], d_nodesX, d_nodesY, d_nodesZ, nodesCount,
                                                  d_elements, k, sumColors, d_D, d_K_x, d_K_y, d_K_value, d_constraints, constraintsCount);
        sumColors += h_colors[k];
        //cout << endl << "!!!" << k << " " << h_colors[k] << endl;
        //break;
        hipDeviceSynchronize();
    }


    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("GPU(ColorStiffnessMatrix) = %.4f ms\n", gpuTime);

//    hipMemcpy(h_K_x, d_K_x, SIZE * sizeof(int), hipMemcpyDeviceToHost);
//    hipMemcpy(h_K_y, d_K_y, SIZE * sizeof(int), hipMemcpyDeviceToHost);
//    hipMemcpy(h_K_value, d_K_value, SIZE * sizeof(float), hipMemcpyDeviceToHost);

//    for (int i = 0; i < SIZE; i++) {
//        //if (h_K_value[i] != 0.0) {
//            cout << h_K_x[i] << " " << h_K_y[i] << " " << h_K_value[i] << endl;
//        //}
//    }

    hipFree(d_nodesX);
    hipFree(d_nodesY);
    hipFree(d_nodesZ);


    CountNonZeroValues<<<1, 1>>> (d_K_x, d_K_y, d_K_value, SIZE, d_nnz);
    hipMemcpy(h_nnz, d_nnz, 1 * sizeof(int), hipMemcpyDeviceToHost);
    cout << "\nNONZERO=" << h_nnz[0] << endl;


//    for (int i = 0; i < SIZE; i++) {
//        ApplyConstraintsCuda<<<(255+constraintsCount)/256, 256>>> (d_K_x, d_K_y, d_K_value, d_constraints, constraintsCount, i);
//    }

//    hipEventRecord(stop, 0);
//    hipEventSynchronize(stop);
//    hipEventElapsedTime(&gpuTime, start, stop);
//    printf("GPU(ApplyConstraints) = %.4f ms\n", gpuTime);


//    CountNonZeroValues<<<1, 1>>> (d_K_x, d_K_y, d_K_value, SIZE, d_nnz);
//    hipMemcpy(h_nnz, d_nnz, 1 * sizeof(int), hipMemcpyDeviceToHost);
//    cout << "NONZERO(After Constraints)=" << h_nnz[0] << endl;


    int *d_ptr, *d_ind, *d_row;
    float *d_data;

    int *h_ptr = new int[3 * nodesCount + 1];
    float *h_data = new float[h_nnz[0]];
    int *h_ind = new int[h_nnz[0]];

    hipMalloc((void**)&d_ptr, (3 * nodesCount + 1) * sizeof(int));
    //hipMalloc((void**)&d_row, h_nnz[0] * sizeof(int));
    hipMalloc((void**)&d_ind, h_nnz[0] * sizeof(int));
    hipMalloc((void**)&d_data, h_nnz[0] * sizeof(float));

    ConvertToCSR<<<1, 1>>> (d_K_x, d_K_y, d_K_value, d_ptr, d_ind, d_data, SIZE, nodesCount);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("GPU(ConvertToCSR) = %.4f ms\n", gpuTime);






    hipFree(d_D);
    hipFree(d_K_value);
    hipFree(d_K_x);
    hipFree(d_K_y);
    hipFree(d_colors);
    hipFree(d_constraints);
    hipFree(d_elements);

//    hipMemcpy(h_ptr, d_ptr, (3 * nodesCount + 1) * sizeof(int), hipMemcpyDeviceToHost);
//    for (int i = 0; i < 3 * nodesCount + 1; i++) {
//        cout << h_ptr[i] << " ";
//    }
//    hipMemcpy(h_ind, d_ind, h_nnz[0] * sizeof(int), hipMemcpyDeviceToHost);
//    for (int i = 0; i < h_nnz[0]; i++) {
//        cout << h_ind[i] << " ";
//    }
//    hipMemcpy(h_data, d_data, h_nnz[0] * sizeof(float), hipMemcpyDeviceToHost);
//    for (int i = 0; i < h_nnz[0]; i++) {
//        cout << h_data[i] << " ";
//    }




    hipsolverSpHandle_t handle;
    hipsolverSpCreate(&handle);
    hipsparseMatDescr_t descr;
    hipsparseCreateMatDescr(&descr);

//    hipMemcpy(d_csrValA, h_csrValA, nnz * sizeof(float), hipMemcpyHostToDevice);
//    hipMemcpy(d_csrRowPtrA, h_csrRowPtrA, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(d_csrColIndA, h_csrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice);



    cout<<"start solving...\n";
    float tol = 1e-16;
    int reorder = 1;
    int singularity = 0;
    hipsolverSpScsrlsvqr(handle, 3 * nodesCount, h_nnz[0], descr, d_data, d_ptr, d_ind, d_b, tol,
                     reorder, d_x, &singularity);

    cout << "end solving...\n";

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("GPU(SOLVER) = %.4f ms\n", gpuTime);

    float *h_x = new float[3 * nodesCount];
    hipMemcpy(h_x, d_x, 3 * nodesCount * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < 3 * nodesCount; i++) {
        cout << h_x[i] << " ";
    }


    hipFree(d_b);
    hipFree(d_x);
    hipFree(d_data);
    hipFree(d_ptr);
    hipFree(d_ind);
    hipFree(d_nnz);
    delete [] h_nnz;

}



void CudaSolve(int *h_csrRowPtrA, int *h_csrColIndA, float *h_csrValA, int n, int nnz, float *h_b, float *h_x) {

    hipsolverSpHandle_t handle;
    hipsolverStatus_t status;
    hipsparseStatus_t status2;

    status = hipsolverSpCreate(&handle);


    hipsparseMatDescr_t descr;
    status2 = hipsparseCreateMatDescr(&descr);



    float* d_csrValA, *d_b, *d_x;
    int* d_csrRowPtrA, *d_csrColIndA;
    hipMalloc((void**)&d_csrValA, nnz * sizeof(float));
    hipMalloc((void**)&d_b, n * sizeof(float));
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_csrRowPtrA, (n + 1) * sizeof(int));
    hipMalloc((void**)&d_csrColIndA, nnz * sizeof(int));


    hipMemcpy(d_csrValA, h_csrValA, nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_csrRowPtrA, h_csrRowPtrA, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrColIndA, h_csrColIndA, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice);



    cout<<"start solving...\n";
    float tol = 1e-16;
    int reorder = 1;
    int singularity = 0;
    status = hipsolverSpScsrlsvqr(handle, n, nnz, descr, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_b, tol,
                     reorder, d_x, &singularity);

    cout<<"end solving...\n";
    hipMemcpy(h_x, d_x, n * sizeof(float), hipMemcpyDeviceToHost);
    //cout<<"singularity = "<<singularity<<"\n";


    hipFree(d_csrValA);
    hipFree(d_csrRowPtrA);
    hipFree(d_csrColIndA);
    hipFree(d_b);
    hipFree(d_x);
    hipsolverSpDestroy(handle);


}

void SortCOO(int *h_cooRows, int *h_cooCols, float *h_cooVals, int n, int nnz) {
    hipsparseHandle_t handle = NULL;
    hipStream_t stream = NULL;


    int *h_P = new int[nnz];

    int *d_cooRows = NULL;
    int *d_cooCols = NULL;
    int *d_P       = NULL;
    float *d_cooVals = NULL;
    float *d_cooVals_sorted = NULL;
    size_t pBufferSizeInBytes = 0;
    void *pBuffer = NULL;

    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipsparseCreate(&handle);
    hipsparseSetStream(handle, stream);

    hipsparseXcoosort_bufferSizeExt(
            handle,
            n,
            n,
            nnz,
            d_cooRows,
            d_cooCols,
            &pBufferSizeInBytes
        );

     printf("pBufferSizeInBytes = %lld bytes \n", (long long)pBufferSizeInBytes);

     hipMalloc( &d_cooRows, sizeof(int)*nnz);
     hipMalloc( &d_cooCols, sizeof(int)*nnz);
        hipMalloc( &d_P      , sizeof(int)*nnz);
        hipMalloc( &d_cooVals, sizeof(float)*nnz);
        hipMalloc( &d_cooVals_sorted, sizeof(float)*nnz);
        hipMalloc( &pBuffer, sizeof(char)* pBufferSizeInBytes);

        hipMemcpy(d_cooRows, h_cooRows, sizeof(int)*nnz   , hipMemcpyHostToDevice);
        hipMemcpy(d_cooCols, h_cooCols, sizeof(int)*nnz   , hipMemcpyHostToDevice);
        hipMemcpy(d_cooVals, h_cooVals, sizeof(float)*nnz, hipMemcpyHostToDevice);
        hipDeviceSynchronize();


        hipsparseCreateIdentityPermutation(handle, nnz, d_P);


        hipsparseXcoosortByRow(
            handle,
            n,
            n,
            nnz,
            d_cooRows,
            d_cooCols,
            d_P,
            pBuffer
        );

        hipsparseSgthr(
            handle,
            nnz,
            d_cooVals,
            d_cooVals_sorted,
            d_P,
            HIPSPARSE_INDEX_BASE_ZERO
        );

        hipDeviceSynchronize();
        hipMemcpy(h_cooRows, d_cooRows, sizeof(int)*nnz   , hipMemcpyDeviceToHost);
        hipMemcpy(h_cooCols, d_cooCols, sizeof(int)*nnz   , hipMemcpyDeviceToHost);
        hipMemcpy(h_P,       d_P      , sizeof(int)*nnz   , hipMemcpyDeviceToHost);
        hipMemcpy(h_cooVals, d_cooVals_sorted, sizeof(float)*nnz, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();


//        printf("sorted coo: \n");
//        for(int j = 0 ; j < nnz; j++){
//            printf("(%d, %d, %f) \n", h_cooRows[j], h_cooCols[j], h_cooVals[j] );
//        }

//        for(int j = 0 ; j < nnz; j++){
//            printf("P[%d] = %d \n", j, h_P[j] );
//        }

}



void setUpDescriptor(hipsparseMatDescr_t& descrA, hipsparseMatrixType_t matrixType, hipsparseIndexBase_t indexBase) {
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, matrixType);
    hipsparseSetMatIndexBase(descrA, indexBase);
}


void setUpDescriptorLU(hipsparseMatDescr_t& descrLU, hipsparseMatrixType_t matrixType, hipsparseIndexBase_t indexBase, hipsparseFillMode_t fillMode, hipsparseDiagType_t diagType) {
    hipsparseCreateMatDescr(&descrLU);
    hipsparseSetMatType(descrLU, matrixType);
    hipsparseSetMatIndexBase(descrLU, indexBase);
    hipsparseSetMatFillMode(descrLU, fillMode);
    hipsparseSetMatDiagType(descrLU, diagType);
}


void memoryQueryLU(csrilu02Info_t& info_A, csrsv2Info_t& info_L, csrsv2Info_t& info_U, hipsparseHandle_t handle, const int N, const int nnz, hipsparseMatDescr_t descrA, hipsparseMatDescr_t descr_L,
    hipsparseMatDescr_t descr_U, float* d_A, int* d_A_RowIndices, int* d_A_ColIndices, hipsparseOperation_t matrixOperation, void** pBuffer) {

    hipsparseCreateCsrilu02Info(&info_A);
    hipsparseCreateCsrsv2Info(&info_L);
    hipsparseCreateCsrsv2Info(&info_U);

    int pBufferSize_M, pBufferSize_L, pBufferSize_U;
//    hipsparseDcsrilu02_bufferSize(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, &pBufferSize_M);
//    hipsparseDcsrsv2_bufferSize(handle, matrixOperation, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, &pBufferSize_L);
//    hipsparseDcsrsv2_bufferSize(handle, matrixOperation, N, nnz, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, &pBufferSize_U);

    int pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));
    hipMalloc((void**)pBuffer, pBufferSize);

}


// ANALYSIS FUNCTION FOR LU DECOMPOSITION
void analysisLUDecomposition(csrilu02Info_t& info_A, csrsv2Info_t& info_L, csrsv2Info_t& info_U, hipsparseHandle_t handle, const int N, const int nnz, hipsparseMatDescr_t descrA, hipsparseMatDescr_t descr_L,
    hipsparseMatDescr_t descr_U, float* d_A, int* d_A_RowIndices, int* d_A_ColIndices, hipsparseOperation_t matrixOperation, hipsparseSolvePolicy_t solvePolicy1, hipsparseSolvePolicy_t solvePolicy2, void* pBuffer) {

    int structural_zero;

//    hipsparseDcsrilu02_analysis(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, solvePolicy1, pBuffer);
    hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(handle, info_A, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status) { printf("A(%d,%d) is missing\n", structural_zero, structural_zero); }

//    hipsparseDcsrsv2_analysis(handle, matrixOperation, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, solvePolicy1, pBuffer);
//    hipsparseDcsrsv2_analysis(handle, matrixOperation, N, nnz, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, solvePolicy2, pBuffer);

}


// COMPUTE LU DECOMPOSITION FOR SPARSE MATRICES

void computeSparseLU(csrilu02Info_t& info_A, hipsparseHandle_t handle, const int N, const int nnz, hipsparseMatDescr_t descrA, float* d_A, int* d_A_RowIndices,
    int* d_A_ColIndices, hipsparseSolvePolicy_t solutionPolicy, void* pBuffer) {

    int numerical_zero;

//    hipsparseDcsrilu02(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, solutionPolicy, pBuffer);
    hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(handle, info_A, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status) { printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero); }

}


void Prepare_CSR(int *rows, int *ptr, int nnz, int n) {
    hipsparseHandle_t    handle;
    hipsparseCreate(&handle);

    int *d_rows;
    hipMalloc(&d_rows, nnz * sizeof(int));
    hipMemcpy(d_rows, rows, nnz * sizeof(int), hipMemcpyHostToDevice);

    int *d_ptr;
    hipMalloc(&d_ptr, (n + 1) * sizeof(int));
    hipMemcpy(d_ptr, ptr, (n + 1) * sizeof(int), hipMemcpyHostToDevice);

    hipsparseXcoo2csr(handle, d_rows, nnz, n, d_ptr, HIPSPARSE_INDEX_BASE_ZERO);

    hipMemcpy(ptr, d_ptr, (n + 1) * sizeof(int), hipMemcpyDeviceToHost);
}

void LU_GPU_SOLVE(int *h_A_RowIndices, int *h_A_ColIndices, float *h_A, int n, int nnz, float *h_x, float *result)
{

    hipsparseHandle_t    handle;

    hipsparseMatDescr_t  descrA = 0;
    hipsparseMatDescr_t  descr_L = 0;
    hipsparseMatDescr_t  descr_U = 0;

    csrilu02Info_t      info_A = 0;
    csrsv2Info_t        info_L = 0;
    csrsv2Info_t        info_U = 0;

    void* pBuffer = 0;

    hipEvent_t start, stop;
    hipsparseCreate(&handle);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float gpuTime = 0.0;

    hipEventRecord(start, 0);


    const int Nrows = n;
    const int Ncols = n;
    const int N = Nrows;


    float* d_x;
    hipMalloc(&d_x, Nrows * sizeof(float));
    hipMemcpy(d_x, h_x, Nrows * sizeof(float), hipMemcpyHostToDevice);



    setUpDescriptor(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ONE);

//    for (int i = 0; i < nnz; i++) {
//        printf("%f ", h_A[i]);
//    }


    for (int i = 0; i < nnz; i++) {
        //h_A_ColIndices[i]++;
        //printf("%d ", h_A_ColIndices[i]);
    }


    float* d_A;
    hipMalloc(&d_A, nnz * sizeof(*d_A));

    int* d_A_RowIndices;
    hipMalloc(&d_A_RowIndices, (Nrows + 1) * sizeof(*d_A_RowIndices));

    int* d_A_ColIndices;
    hipMalloc(&d_A_ColIndices, nnz * sizeof(*d_A_ColIndices));

    hipMemcpy(d_A, h_A, nnz * sizeof(*h_A), hipMemcpyHostToDevice);
    hipMemcpy(d_A_RowIndices, h_A_RowIndices, (Nrows + 1) * sizeof(*h_A_RowIndices), hipMemcpyHostToDevice);
    hipMemcpy(d_A_ColIndices, h_A_ColIndices, nnz * sizeof(*h_A_ColIndices), hipMemcpyHostToDevice);


    hipMemcpy(h_A, d_A, nnz * sizeof(*h_A), hipMemcpyDeviceToHost);

    setUpDescriptorLU(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ONE, HIPSPARSE_FILL_MODE_LOWER, HIPSPARSE_DIAG_TYPE_UNIT);
    setUpDescriptorLU(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ONE, HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);


    memoryQueryLU(info_A, info_L, info_U, handle, N, nnz, descrA, descr_L, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, HIPSPARSE_OPERATION_NON_TRANSPOSE, &pBuffer);


    analysisLUDecomposition(info_A, info_L, info_U, handle, N, nnz, descrA, descr_L, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_SOLVE_POLICY_NO_LEVEL,HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);


    computeSparseLU(info_A, handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);


    float* d_z;
    hipMalloc(&d_z, N * sizeof(float));

    const float alpha = 1.;
//    hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, &alpha, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, d_x, d_z, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);


    float* d_y;
    hipMalloc(&d_y, N * sizeof(float));

//    hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, &alpha, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, d_z, d_y, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);


    float* h_y = (float*)malloc(Ncols * sizeof(float));
    hipMemcpy(h_x, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("GPU time = %.4f \n", gpuTime);

    printf("\n\nFinal result\n");
    for (int i = 0; i < N; i++) {
        printf("%f ", h_x[i]);
    }
}




///////////////////////////////////////////////////////
__global__
void cgm_gpu(float *z_k, float *r_k, float *Az,
             int* x, int *y, float *data, float *b,
             float *x_k, const int n, int sparse_size, float *partialSum) {

    float mf = 0.0, alpha, beta, eps = 0.00001, Spz, Spr, Spr1;

    int tx = threadIdx.x;
    int i = tx + blockIdx.x * blockDim.x;

    if (i < n) {
        partialSum[tx] = b[i] * b[i];
    }

    int stride;
    for (stride = blockDim.x/2; stride > 0;  stride >>= 1) {
        __syncthreads();
        if (tx < stride) {
            partialSum[tx] += partialSum[tx + stride];
        }
    }

    if (tx == 0) {
        //b[blockIdx.x] = partialSum[tx];
        mf = partialSum[tx];
    }



    x_k[i] = 0.2;
    Az[i] = 0.0;

    if (i < sparse_size) {
        Az[x[i]] += data[i] * x_k[y[i]];
    }
    r_k[i] = b[i] - Az[i];
    z_k[i] = r_k[i];

    //do{
        Spz=0.0;
        Spr=0.0;
        Az[i] = 0.0;
        if (i < sparse_size) {
            Az[x[i]] += data[i] * z_k[y[i]];
        }
        //Spz
        if (i < n) {
            partialSum[tx] = Az[i] * z_k[i];
        }

        for (stride = blockDim.x/2; stride > 0;  stride >>= 1) {
            __syncthreads();
            if (tx < stride) {
                partialSum[tx] += partialSum[tx + stride];
            }
        }
        if (tx == 0) {
            //b[blockIdx.x] = partialSum[tx];
            Spz = partialSum[tx];
            printf("Spz=%f\n", Spz);
        }
        //Spr
        if (i < n) {
            partialSum[tx] = r_k[i] * r_k[i];
        }

        for (stride = blockDim.x/2; stride > 0;  stride >>= 1) {
            __syncthreads();
            if (tx < stride) {
                partialSum[tx] += partialSum[tx + stride];
            }
        }

        if (tx == 0) {
            //b[blockIdx.x] = partialSum[tx];
            Spr = partialSum[tx];
            printf("Spr=%f\n", Spr);
        }
        ////
        alpha = Spr / Spz;
        Spr1 = 0.0;
        x_k[i] += alpha * z_k[i];
        r_k[i] -= alpha * Az[i];
        //Spr1
        if (i < n) {
            partialSum[tx] = r_k[i] * r_k[i];
            printf("%f ", r_k[i]);
        }

        for (stride = blockDim.x/2; stride > 0;  stride >>= 1) {
            __syncthreads();
            if (tx < stride) {
                partialSum[tx] += partialSum[tx + stride];
            }
        }

        if (tx == 0) {
            //b[blockIdx.x] = partialSum[tx];
            Spr1 = partialSum[tx];
            printf("Spr1=%f\n", Spr1);
        }
        ////
        beta = Spr1 / Spr;
        z_k[i] = r_k[i] + beta * z_k[i];


   // } while (Spr1 / mf > eps * eps);

    if (i == 0)
        printf("GPU CGM SUCCESS\n");
}

void callCGM_GPU(int *x, int *y, float *data, float *b, float *x_k, int n, int sparse_size) {
    float *z_k, *r_k, *Az;
    float *d_z_k, *d_r_k, *d_Az, *d_data, *d_b, *d_x_k, *partialSum;
    int *d_x, *d_y;
    z_k = (float*)malloc(n * sizeof(float));
    r_k = (float*)malloc(n * sizeof(float));
    Az = (float*)malloc(n * sizeof(float));


    hipMalloc(&d_z_k, n * sizeof(float));
    hipMalloc(&d_r_k, n * sizeof(float));
    hipMalloc(&d_Az, n * sizeof(float));
    hipMalloc(&d_x, sparse_size * sizeof(int));
    hipMalloc(&d_y, sparse_size * sizeof(int));
    hipMalloc(&d_data, sparse_size * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));
    hipMalloc(&d_x_k, n * sizeof(float));
    hipMalloc(&partialSum, n * sizeof(float));

    hipMemcpy(d_z_k, z_k, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_r_k, r_k, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Az, Az, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, sparse_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sparse_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_data, data, sparse_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x_k, x_k, n * sizeof(float), hipMemcpyHostToDevice);

    cgm_gpu<<<1, n>>>(d_z_k, d_r_k, d_Az, d_x, d_y, d_data, d_b, d_x_k, n, sparse_size, partialSum);

    hipMemcpy(x_k, d_x_k, n * sizeof(float), hipMemcpyDeviceToHost);


//    hipFree(d_x);
//    hipFree(d_y);
//    free(x);
//    free(y);
}



__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void callCudaKernel()
{
  int N = 1<<8;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = max(maxError, abs(y[i]-4.0f));
    printf("%f ", y[i]);
  }
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}

